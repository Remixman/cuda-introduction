
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

#define N 3000000
#define BLOCKSIZE 256

__global__ void moving_average(float *in, float *out) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < N-2) {
    out[i] = (in[i] + in[i+1] + in[i+2]) / 3.0;
  }
}

int main() {
  
  float *in, *out;
  float *d_in, *d_out;
  size_t vecSize = N * sizeof(float);

  in = (float*)malloc(vecSize);
  out = (float*)malloc((N-2) * sizeof(float));

  // Allocate device memory for vector a, b and c
  hipMalloc((void**)&d_in, vecSize);
  hipMalloc((void**)&d_out, (N-2) * sizeof(float));

  // Transfer data from host to device
  hipMemcpy(d_in, in, vecSize, hipMemcpyHostToDevice);

  // Call kernel
  int threadsPerBlock = BLOCKSIZE;
  int numBlocks = ceil((N-2) * 1.0 / threadsPerBlock);
  moving_average<<<numBlocks, threadsPerBlock>>>(d_in, d_out);

  // Transfer data from device to host
  hipMemcpy(out, d_out, (N-2) * sizeof(float), hipMemcpyDeviceToHost);

  // Deallocate device memory
  hipFree(d_in);
  hipFree(d_out);

  free(in); free(out);
  
  return 0;
}